#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <mpi.h>

#include "l7/l7.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void
init_int_array(const int  num_indices_have, // 0
               const int  my_start_id,      // 1
               int *array)            // 2
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   if (i >= num_indices_have) return;

   array[i] = i + my_start_id;
}

__global__ void
init_double_array(const int  num_indices_have, // 0
                  const int  my_start_id,      // 1
                  double *array)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   if (i >= num_indices_have) return;

   array[i] = i + my_start_id;
}

__global__ void
init_short_array(const int  num_indices_have, // 0
                 const int  my_start_id,      // 1
                 short *array)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   if (i >= num_indices_have) return;

   array[i] = i + my_start_id;
}


__global__ void
init_char_array(const int  num_indices_have, // 0
                const int  my_start_id,      // 1
                char *array)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   if (i >= num_indices_have) return;

   array[i] = i + my_start_id;
}

extern "C"
void initialize_data_cuda(void **odata, int nowned, int nremote, int typesize, int my_start_index)
{
   /*
    * Allocate data arrays on device and wait for initialization to complete
    */
   unsigned long data_size;
   data_size = nowned + nremote;
   switch (typesize) {
   case 1:
      hipMalloc(odata, sizeof(char) * data_size);
      init_char_array<<<(nowned + 255)/256, 256>>>(nowned, my_start_index, *(char **)odata);
      break;
   case 2:
      hipMalloc(odata, sizeof(short) * data_size);
      init_short_array<<<(nowned + 255)/256, 256>>>(nowned, my_start_index, *(short **)odata);
      break;
   case 4:
      hipMalloc(odata, sizeof(int) * data_size);
      init_int_array<<<(nowned + 255)/256, 256>>>(nowned, my_start_index, *(int **)odata);
      break;
   case 8:
      hipMalloc(odata, sizeof(double) * data_size);
      init_double_array<<<(nowned + 255)/256, 256>>>(nowned, my_start_index, *(double **)odata);
      break;
   }
   hipDeviceSynchronize();
   return;
}
